#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#define BLOCK_WIDTH 2
#define TILE_WIDTH 2
#define WIDTH 2
__device__ int getTid() {
	int blockSkip = (blockIdx.y * gridDim.x * blockDim.x * blockDim.y);
	int rowSkip = (threadIdx.y * gridDim.x * blockDim.x);
	int rowDisplacement = (blockIdx.x * blockDim.x) + threadIdx.x;
	int tid = blockSkip + rowSkip + rowDisplacement;
	return tid;
}
__global__ void MatAddElementThread(int *a, int *b, int *d) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;
    for (int k = 0; k < WIDTH; k++) {
        sum += a[row * WIDTH + k] * b[k * WIDTH + col];
    }

    d[row * WIDTH + col] = sum;
}

int main() {
	int *matA, *matB, *matSum;
	int *da, *db, *dc;
	printf("\n== Enter elements of Matrix A==\n");
	matA = (int*)malloc(sizeof(int) * WIDTH * WIDTH);
	for(int i = 0; i < WIDTH * WIDTH; i++)
	{
	scanf("%d", &matA[i]);
	}
	printf("\n== Enter elements of Matrix B==\n");
	matB = (int*)malloc(sizeof(int) * WIDTH * WIDTH);
	for(int i = 0; i < WIDTH * WIDTH; i++)
	{
	scanf("%d", &matB[i]);
	}
	matSum = (int*)malloc(sizeof(int) * WIDTH * WIDTH);
	hipMalloc((void **) &da, sizeof(int) * WIDTH * WIDTH);
	hipMalloc((void **) &db, sizeof(int) * WIDTH * WIDTH);
	hipMalloc((void **) &dc, sizeof(int) * WIDTH * WIDTH);
	hipMemcpy(da, matA, sizeof(int) * WIDTH *WIDTH, hipMemcpyHostToDevice);
	hipMemcpy(db, matB, sizeof(int) * WIDTH *WIDTH, hipMemcpyHostToDevice);
	int NumBlocks = WIDTH / BLOCK_WIDTH;
	dim3 grid_conf (NumBlocks, NumBlocks);
	dim3 block_conf (BLOCK_WIDTH, BLOCK_WIDTH);
	MatAddElementThread<<<grid_conf, block_conf>>>(da, db, dc);
	hipMemcpy(matSum,dc,sizeof(int)* WIDTH *WIDTH,hipMemcpyDeviceToHost);
	printf("\n-=Result=-\n");
	printf("----------------------\n");
	for (int i = 0; i < WIDTH; i++ ) {
	for (int j = 0; j < WIDTH; j++) {
	printf("%6d ", matSum[i * WIDTH + j]);
	}
	printf("\n");
	}
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	free(matA);
	free(matB);
	free(matSum);
	return 0;
}