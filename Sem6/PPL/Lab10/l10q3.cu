
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void csr_matrix_vector_multiplication(int *row_ptr, int *col_index, float *values, float *vector, float *result, int rows) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(row < rows) {
        float sum = 0.0;
        for(int j = row_ptr[row]; j < row_ptr[row + 1]; j++) {
            sum += values[j] * vector[col_index[j]];
        }
        result[row] = sum;
    }
}

int main() {
    int rows = 3;
    int cols = 3;
    
    // CSR format for a 3x3 matrix
    int row_ptr[4] = {0, 2, 3, 4};
    int col_index[4] = {0, 2, 1, 2};
    float values[4] = {1.0, 2.0, 3.0, 4.0};
    
    float vector[3] = {1.0, 2.0, 3.0};
    float result[3];
    
    int *d_row_ptr, *d_col_index;
    float *d_values, *d_vector, *d_result;
    
    hipMalloc(&d_row_ptr, 4 * sizeof(int));
    hipMemcpy(d_row_ptr, row_ptr, 4 * sizeof(int), hipMemcpyHostToDevice);
    
    hipMalloc(&d_col_index, 4 * sizeof(int));
    hipMemcpy(d_col_index, col_index, 4 * sizeof(int), hipMemcpyHostToDevice);
    
    hipMalloc(&d_values, 4 * sizeof(float));
    hipMemcpy(d_values, values, 4 * sizeof(float), hipMemcpyHostToDevice);
    
    hipMalloc(&d_vector, 3 * sizeof(float));
    hipMemcpy(d_vector, vector, 3 * sizeof(float), hipMemcpyHostToDevice);
    
    hipMalloc(&d_result, 3 * sizeof(float));
    
    int block_size = 32;
    int num_blocks = (rows + block_size - 1) / block_size;
    
    csr_matrix_vector_multiplication<<<num_blocks, block_size>>>(d_row_ptr, d_col_index, d_values, d_vector, d_result, rows);
    
    hipMemcpy(result, d_result, 3 * sizeof(float), hipMemcpyDeviceToHost);
    
    for(int i = 0; i < rows; i++) {
        printf("Result[%d] = %.2f", i, result[i]);
    }
    
    hipFree(d_row_ptr);
    hipFree(d_col_index);
    hipFree(d_values);
    hipFree(d_vector);
    hipFree(d_result);
    
    return 0;
}
