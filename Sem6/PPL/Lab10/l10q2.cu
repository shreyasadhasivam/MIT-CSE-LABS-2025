

#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE 5

__global__ void convolution_2d(int *input, int *mask, int *output) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < SIZE && col < SIZE) {
        int sum = 0;
        for (int i = 0; i < SIZE; i++) {
            for (int j = 0; j < SIZE; j++) {
                int idx = (row + i - SIZE/2) * SIZE + (col + j - SIZE/2);
                if (idx >= 0 && idx < SIZE*SIZE) {
                    sum += input[idx] * mask[i*SIZE+j];
                }
            }
        }
        output[row * SIZE + col] = sum;
    }
}

int main() {
    int input[SIZE*SIZE] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24, 25};
    int mask[SIZE*SIZE] = {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1};
    int output[SIZE*SIZE] = {0};

    int *d_input, *d_mask, *d_output;

    hipMalloc(&d_input, SIZE*SIZE*sizeof(int));
    hipMalloc(&d_mask, SIZE*SIZE*sizeof(int));
    hipMalloc(&d_output, SIZE*SIZE*sizeof(int));

    hipMemcpy(d_input, input, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((SIZE + 31)/32, (SIZE + 31)/32);

    convolution_2d<<<numBlocks, threadsPerBlock>>>(d_input, d_mask, d_output);

    hipMemcpy(output, d_output, SIZE*SIZE*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < SIZE; i++) {
        for (int j = 0; j < SIZE; j++) {
            printf("%d ", output[i*SIZE+j]);
        }
        printf("\n");
    }

    hipFree(d_input);
    hipFree(d_mask);
    hipFree(d_output);

    return 0;
}
