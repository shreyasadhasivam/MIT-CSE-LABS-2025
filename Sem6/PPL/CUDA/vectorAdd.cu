
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 
    float *d_a, *d_b, *d_out;
    float N;

    printf("Enter number of elements:");
    scanf("%f",&N);
    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = i;
        b[i] = i+5;
    }

    //allocate device memory
    hipMalloc((void**)&d_a,sizeof(float)*N);
    hipMalloc((void**)&d_b,sizeof(float)*N);
    hipMalloc((void**)&d_out,sizeof(float)*N);


    //transfer data from host to device memory
    hipMemcpy(d_a,a,sizeof(float)*N,hipMemcpyHostToDevice);
    hipMemcpy(d_a,a,sizeof(float)*N,hipMemcpyHostToDevice);

    // Main function
    vector_add<<<1,1>>>(out, a, b, N);

    //transfer data back to host memory
    hipMemcpy(out,d_out,sizeof(float)*N,hipMemcpyDeviceToHost);
    printf("A vector: ");
    for(int i=0;i<N;i++)
    {
        printf("%.2f ",a[i]);
    }
    printf("B vector: ");
    for(int i=0;i<N;i++)
    {
        printf("%.2f ",b[i]);
    }
    printf("Resultant vector: ");
    for(int i=0;i<N;i++)
    {
        printf("%.2f ",out[i]);
    }

    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    hipFree(d_a);
    free(a);
}