#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 3  // Size of matrices

__global__ void addMatricesRow(int* A, int* B, int* C) {
    int i = blockIdx.x;  // Each row computed by one thread
    for (int j = 0; j < N; j++) {
        C[i * N + j] = A[i * N + j] + B[i * N + j];
    }
}

__global__ void addMatricesColumn(int* A, int* B, int* C) {
    int j = blockIdx.x;  // Each column computed by one thread
    for (int i = 0; i < N; i++) {
        C[i * N + j] = A[i * N + j] + B[i * N + j];
    }
}

__global__ void addMatricesElement(int* A, int* B, int* C) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Each element computed by one thread
    int i = idx / N;
    int j = idx % N;
    if (i < N && j < N) {
        C[i * N + j] = A[i * N + j] + B[i * N + j];
    }
}

int main() {
    int A[N * N] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    int B[N * N] = {9, 8, 7, 6, 5, 4, 3, 2, 1};
    int C[N * N] = {0};

    int *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * N * sizeof(int));
    hipMalloc((void**)&d_B, N * N * sizeof(int));
    hipMalloc((void**)&d_C, N * N * sizeof(int));
    hipMemcpy(d_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Add matrices using rows
    addMatricesRow<<<N, 1>>>(d_A, d_B, d_C);
    printf("\nRows\n");
    hipMemcpy(C, d_C, N * N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ",C[i * N + j]);
        }
        printf("\n");
    }
    // Add matrices using columns
    printf("\nColumns\n");
    addMatricesColumn<<<N, 1>>>(d_A, d_B, d_C);
    hipMemcpy(C, d_C, N * N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ",C[i * N + j]);
        }
        printf("\n");
    }

    // Add matrices using elements
    printf("\nElements\n");
    addMatricesElement<<<(N * N + 255) / 256, 256>>>(d_A, d_B, d_C);

    hipMemcpy(C, d_C, N * N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ",C[i * N + j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
