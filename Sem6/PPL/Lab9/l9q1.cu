
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void rowOperations(int *matrix, int rows, int cols) {
    int row = blockIdx.x;
    int col = threadIdx.x;

    int index = row * cols + col;

    if (row == 0) {
        // do nothing, keep original values
    } else if (row % 2 == 1) {
        // square elements
        matrix[index] = matrix[index] * matrix[index];
    } else {
        // cube elements
        matrix[index] = matrix[index] * matrix[index] * matrix[index];
    }
}

int main() {
    int rows, cols;

    printf("Enter number of rows: ");
    scanf("%d", &rows);
    printf("Enter number of columns: ");
    scanf("%d", &cols);

    int size = rows * cols * sizeof(int);
    int *matrix, *d_matrix;

    matrix = (int *) malloc(size);
    hipMalloc(&d_matrix, size);

    printf("Enter elements of matrix (%dx%d): \n", rows, cols);
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            scanf("%d", &matrix[i * cols + j]);
        }
    }

    hipMemcpy(d_matrix, matrix, size, hipMemcpyHostToDevice);

    dim3 grid(rows, 1);
    dim3 block(cols, 1);
    rowOperations<<<grid, block>>>(d_matrix, rows, cols);

    hipMemcpy(matrix, d_matrix, size, hipMemcpyDeviceToHost);

    printf("\nMatrix after row operations: \n");
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%d ", matrix[i * cols + j]);
        }
        printf("\n");
    }

    free(matrix);
    hipFree(d_matrix);

    return 0;
}
